
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define CUDA_ERROR_EXIT(str) do{\
    hipError_t err = hipGetLastError();\
    if( err != hipSuccess){\
        printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
        exit(-1);\
    }\
}while(0);

__global__ void Xor(int *dX,int num)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(num%2==0){
        if(i>num/2||i==0)
        return;
        dX[i]=dX[i]^dX[i+num/2];
    }
    if(num%2!=0){
        if(i>num/2||i==0)
        return;
        dX[i]=dX[i]^dX[i+num/2+1];
    }
}

int main(int argc, char **argv)
{
    int ctr;
    int *hX,*S,*dX;

    /*Allocate memory on the host (CPU) */

    if(argc!=3){
        printf("Incorrect number of arguments\n");
        exit(1);
    }

    int n = atoi(argv[1]);
    int seed = atoi(argv[2]);
    int size = (n+1) * sizeof(int);

    hX = (int *) malloc(size);
    if(!hX){
        perror("malloc");
        exit(-1);
    }

    S= (int *)malloc(2*sizeof(int));
    //printf("%d\n",n);
    srand(seed);
    hX[0]=0;
    for(ctr=1; ctr <= n; ++ctr){
        hX[ctr] = random()/*%10000*/;
        //printf("%d\n",hX[ctr]);
    }

    /*Allocate memory on the device (GPU) */

    hipMalloc(&dX,  size);
    CUDA_ERROR_EXIT("hipMalloc");

    /*Copy hX --> dX */

    hipMemcpy(dX, hX, size, hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("memcpy1");

    int blocks;

    /*Invoke the kernel*/

    int num=n;
    while(num>1){
        blocks=(num+1023)>>10;
        //printf("num= %d\n",num);
        Xor<<<blocks, 1024>>>(dX, num);
        CUDA_ERROR_EXIT("kernel invocation");
        num=num/2+num%2;
    }

    /*Copy back results*/
    hipMemcpy(S, dX, 2*sizeof(int), hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");

    printf("%d\n", S[1]);

    free(hX);
    free(S);
    hipFree(dX);
}
